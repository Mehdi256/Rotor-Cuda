#include "hip/hip_runtime.h"
/*
 * This file is part of the VanitySearch distribution (https://github.com/JeanLucPons/VanitySearch).
 * Copyright (c) 2019 Jean Luc PONS.
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, version 3.
 *
 * This program is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU
 * General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program. If not, see <http://www.gnu.org/licenses/>.
*/

#include "GPUEngine.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#include <stdint.h>
#include "../hash/sha256.h"
#include "../hash/ripemd160.h"
#include "../Timer.h"

#include "GPUMath.h"
#include "GPUHash.h"
#include "GPUBase58.h"
#include "GPUCompute.h"

// ---------------------------------------------------------------------------------------
#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )

inline void __cudaSafeCall(hipError_t err, const char* file, const int line)
{
	if (hipSuccess != err)
	{
		fprintf(stderr, "cudaSafeCall() failed at %s:%i : %s\n", file, line, hipGetErrorString(err));
		exit(-1);
	}
	return;
}

// ---------------------------------------------------------------------------------------

// mode multiple addresses
__global__ void compute_keys_mode_ma(uint32_t mode, uint8_t* bloomLookUp, int BLOOM_BITS, uint8_t BLOOM_HASHES,
	uint64_t* keys, uint32_t maxFound, uint32_t* found)
{

	int xPtr = (blockIdx.x * blockDim.x) * 8;
	int yPtr = xPtr + 4 * blockDim.x;
	ComputeKeysSEARCH_MODE_MA(mode, keys + xPtr, keys + yPtr, bloomLookUp, BLOOM_BITS, BLOOM_HASHES, maxFound, found);

}

__global__ void compute_keys_comp_mode_ma(uint32_t mode, uint8_t* bloomLookUp, int BLOOM_BITS, uint8_t BLOOM_HASHES, uint64_t* keys,
	uint32_t maxFound, uint32_t* found)
{

	int xPtr = (blockIdx.x * blockDim.x) * 8;
	int yPtr = xPtr + 4 * blockDim.x;
	ComputeKeysSEARCH_MODE_MA(mode, keys + xPtr, keys + yPtr, bloomLookUp, BLOOM_BITS, BLOOM_HASHES, maxFound, found);

}

// mode single address
__global__ void compute_keys_mode_sa(uint32_t mode, uint32_t* hash160, uint64_t* keys, uint32_t maxFound, uint32_t* found)
{

	int xPtr = (blockIdx.x * blockDim.x) * 8;
	int yPtr = xPtr + 4 * blockDim.x;
	ComputeKeysSEARCH_MODE_SA(mode, keys + xPtr, keys + yPtr, hash160, maxFound, found);

}

__global__ void compute_keys_comp_mode_sa(uint32_t mode, uint32_t* hash160, uint64_t* keys, uint32_t maxFound, uint32_t* found)
{

	int xPtr = (blockIdx.x * blockDim.x) * 8;
	int yPtr = xPtr + 4 * blockDim.x;
	ComputeKeysSEARCH_MODE_SA(mode, keys + xPtr, keys + yPtr, hash160, maxFound, found);

}

// mode multiple x points
__global__ void compute_keys_comp_mode_mx(uint32_t mode, uint8_t* bloomLookUp, int BLOOM_BITS, uint8_t BLOOM_HASHES, uint64_t* keys,
	uint32_t maxFound, uint32_t* found)
{

	int xPtr = (blockIdx.x * blockDim.x) * 8;
	int yPtr = xPtr + 4 * blockDim.x;
	ComputeKeysSEARCH_MODE_MX(mode, keys + xPtr, keys + yPtr, bloomLookUp, BLOOM_BITS, BLOOM_HASHES, maxFound, found);

}

// mode single x point
__global__ void compute_keys_comp_mode_sx(uint32_t mode, uint32_t* xpoint, uint64_t* keys, uint32_t maxFound, uint32_t* found)
{

	int xPtr = (blockIdx.x * blockDim.x) * 8;
	int yPtr = xPtr + 4 * blockDim.x;
	ComputeKeysSEARCH_MODE_SX(mode, keys + xPtr, keys + yPtr, xpoint, maxFound, found);

}

// ---------------------------------------------------------------------------------------
// ethereum

__global__ void compute_keys_mode_eth_ma(uint8_t* bloomLookUp, int BLOOM_BITS, uint8_t BLOOM_HASHES, uint64_t* keys,
	uint32_t maxFound, uint32_t* found)
{

	int xPtr = (blockIdx.x * blockDim.x) * 8;
	int yPtr = xPtr + 4 * blockDim.x;
	ComputeKeysSEARCH_ETH_MODE_MA(keys + xPtr, keys + yPtr, bloomLookUp, BLOOM_BITS, BLOOM_HASHES, maxFound, found);

}

__global__ void compute_keys_mode_eth_sa(uint32_t* hash, uint64_t* keys, uint32_t maxFound, uint32_t* found)
{

	int xPtr = (blockIdx.x * blockDim.x) * 8;
	int yPtr = xPtr + 4 * blockDim.x;
	ComputeKeysSEARCH_ETH_MODE_SA(keys + xPtr, keys + yPtr, hash, maxFound, found);

}

// ---------------------------------------------------------------------------------------

using namespace std;

int _ConvertSMVer2Cores(int major, int minor)
{

	// Defines for GPU Architecture types (using the SM version to determine
	// the # of cores per SM
	typedef struct {
		int SM;  // 0xMm (hexidecimal notation), M = SM Major version,
		// and m = SM minor version
		int Cores;
	} sSMtoCores;

	sSMtoCores nGpuArchCoresPerSM[] = {
		{0x20, 32}, // Fermi Generation (SM 2.0) GF100 class
		{0x21, 48}, // Fermi Generation (SM 2.1) GF10x class
		{0x30, 192},
		{0x32, 192},
		{0x35, 192},
		{0x37, 192},
		{0x50, 128},
		{0x52, 128},
		{0x53, 128},
		{0x60,  64},
		{0x61, 128},
		{0x62, 128},
		{0x70,  64},
		{0x72,  64},
		{0x75,  64},
		{0x80,  64},
		{0x86, 128},
		{-1, -1}
	};

	int index = 0;

	while (nGpuArchCoresPerSM[index].SM != -1) {
		if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor)) {
			return nGpuArchCoresPerSM[index].Cores;
		}

		index++;
	}

	return 0;

}

// ----------------------------------------------------------------------------

GPUEngine::GPUEngine(Secp256K1* secp, int nbThreadGroup, int nbThreadPerGroup, int gpuId, uint32_t maxFound,
	int searchMode, int compMode, int coinType, int64_t BLOOM_SIZE, uint64_t BLOOM_BITS,
	uint8_t BLOOM_HASHES, const uint8_t* BLOOM_DATA, uint8_t* DATA, uint64_t TOTAL_COUNT, bool rKey)
{

	// Initialise CUDA
	this->nbThreadPerGroup = nbThreadPerGroup;
	this->searchMode = searchMode;
	this->compMode = compMode;
	this->coinType = coinType;
	this->rKey = rKey;

	this->BLOOM_SIZE = BLOOM_SIZE;
	this->BLOOM_BITS = BLOOM_BITS;
	this->BLOOM_HASHES = BLOOM_HASHES;
	this->DATA = DATA;
	this->TOTAL_COUNT = TOTAL_COUNT;

	initialised = false;

	int deviceCount = 0;
	CudaSafeCall(hipGetDeviceCount(&deviceCount));

	// This function call returns 0 if there are no CUDA capable devices.
	if (deviceCount == 0) {
		printf("GPUEngine: There are no available device(s) that support CUDA\n");
		return;
	}

	CudaSafeCall(hipSetDevice(gpuId));

	hipDeviceProp_t deviceProp;
	CudaSafeCall(hipGetDeviceProperties(&deviceProp, gpuId));

	if (nbThreadGroup == -1)
		nbThreadGroup = deviceProp.multiProcessorCount * 8;

	this->nbThread = nbThreadGroup * nbThreadPerGroup;
	this->maxFound = maxFound;
	this->outputSize = (maxFound * ITEM_SIZE_A + 4);
	if (this->searchMode == (int)SEARCH_MODE_MX)
		this->outputSize = (maxFound * ITEM_SIZE_X + 4);

	char tmp[512];
	sprintf(tmp, "GPU #%d %s (%dx%d cores) Grid(%dx%d)",
		gpuId, deviceProp.name, deviceProp.multiProcessorCount,
		_ConvertSMVer2Cores(deviceProp.major, deviceProp.minor),
		nbThread / nbThreadPerGroup,
		nbThreadPerGroup);
	deviceName = std::string(tmp);

	// Prefer L1 (We do not use __shared__ at all)
	CudaSafeCall(hipDeviceSetCacheConfig(hipFuncCachePreferL1));

	size_t stackSize = 49152;
	CudaSafeCall(hipDeviceSetLimit(hipLimitStackSize, stackSize));

	// Allocate memory
	CudaSafeCall(hipMalloc((void**)&inputKey, nbThread * 32 * 2));
	CudaSafeCall(hipHostAlloc(&inputKeyPinned, nbThread * 32 * 2, hipHostMallocWriteCombined | hipHostMallocMapped));

	CudaSafeCall(hipMalloc((void**)&outputBuffer, outputSize));
	CudaSafeCall(hipHostAlloc(&outputBufferPinned, outputSize, hipHostMallocWriteCombined | hipHostMallocMapped));

	CudaSafeCall(hipMalloc((void**)&inputBloomLookUp, BLOOM_SIZE));
	CudaSafeCall(hipHostAlloc(&inputBloomLookUpPinned, BLOOM_SIZE, hipHostMallocWriteCombined | hipHostMallocMapped));

	memcpy(inputBloomLookUpPinned, BLOOM_DATA, BLOOM_SIZE);

	CudaSafeCall(hipMemcpy(inputBloomLookUp, inputBloomLookUpPinned, BLOOM_SIZE, hipMemcpyHostToDevice));
	CudaSafeCall(hipHostFree(inputBloomLookUpPinned));
	inputBloomLookUpPinned = NULL;

	// generator table
	InitGenratorTable(secp);


	CudaSafeCall(hipGetLastError());

	compMode = SEARCH_COMPRESSED;
	initialised = true;

}

// ----------------------------------------------------------------------------

GPUEngine::GPUEngine(Secp256K1* secp, int nbThreadGroup, int nbThreadPerGroup, int gpuId, uint32_t maxFound,
	int searchMode, int compMode, int coinType, const uint32_t* hashORxpoint, bool rKey)
{

	// Initialise CUDA
	this->nbThreadPerGroup = nbThreadPerGroup;
	this->searchMode = searchMode;
	this->compMode = compMode;
	this->coinType = coinType;
	this->rKey = rKey;

	initialised = false;

	int deviceCount = 0;
	CudaSafeCall(hipGetDeviceCount(&deviceCount));

	// This function call returns 0 if there are no CUDA capable devices.
	if (deviceCount == 0) {
		printf("GPUEngine: There are no available device(s) that support CUDA\n");
		return;
	}

	CudaSafeCall(hipSetDevice(gpuId));

	hipDeviceProp_t deviceProp;
	CudaSafeCall(hipGetDeviceProperties(&deviceProp, gpuId));

	if (nbThreadGroup == -1)
		nbThreadGroup = deviceProp.multiProcessorCount * 8;

	this->nbThread = nbThreadGroup * nbThreadPerGroup;
	this->maxFound = maxFound;
	this->outputSize = (maxFound * ITEM_SIZE_A + 4);
	if (this->searchMode == (int)SEARCH_MODE_SX)
		this->outputSize = (maxFound * ITEM_SIZE_X + 4);

	char tmp[512];
	sprintf(tmp, "GPU #%d %s (%dx%d cores) Grid(%dx%d)",
		gpuId, deviceProp.name, deviceProp.multiProcessorCount,
		_ConvertSMVer2Cores(deviceProp.major, deviceProp.minor),
		nbThread / nbThreadPerGroup,
		nbThreadPerGroup);
	deviceName = std::string(tmp);

	// Prefer L1 (We do not use __shared__ at all)
	CudaSafeCall(hipDeviceSetCacheConfig(hipFuncCachePreferL1));

	size_t stackSize = 49152;
	CudaSafeCall(hipDeviceSetLimit(hipLimitStackSize, stackSize));

	// Allocate memory
	CudaSafeCall(hipMalloc((void**)&inputKey, nbThread * 32 * 2));
	CudaSafeCall(hipHostAlloc(&inputKeyPinned, nbThread * 32 * 2, hipHostMallocWriteCombined | hipHostMallocMapped));

	CudaSafeCall(hipMalloc((void**)&outputBuffer, outputSize));
	CudaSafeCall(hipHostAlloc(&outputBufferPinned, outputSize, hipHostMallocWriteCombined | hipHostMallocMapped));

	int K_SIZE = 5;
	if (this->searchMode == (int)SEARCH_MODE_SX)
		K_SIZE = 8;

	CudaSafeCall(hipMalloc((void**)&inputHashORxpoint, K_SIZE * sizeof(uint32_t)));
	CudaSafeCall(hipHostAlloc(&inputHashORxpointPinned, K_SIZE * sizeof(uint32_t), hipHostMallocWriteCombined | hipHostMallocMapped));

	memcpy(inputHashORxpointPinned, hashORxpoint, K_SIZE * sizeof(uint32_t));

	CudaSafeCall(hipMemcpy(inputHashORxpoint, inputHashORxpointPinned, K_SIZE * sizeof(uint32_t), hipMemcpyHostToDevice));
	CudaSafeCall(hipHostFree(inputHashORxpointPinned));
	inputHashORxpointPinned = NULL;

	// generator table
	InitGenratorTable(secp);


	CudaSafeCall(hipGetLastError());

	compMode = SEARCH_COMPRESSED;
	initialised = true;

}

// ----------------------------------------------------------------------------

void GPUEngine::InitGenratorTable(Secp256K1* secp)
{

	// generator table
	uint64_t* _2GnxPinned;
	uint64_t* _2GnyPinned;

	uint64_t* GxPinned;
	uint64_t* GyPinned;

	uint64_t size = (uint64_t)GRP_SIZE;

	CudaSafeCall(hipMalloc((void**)&__2Gnx, 4 * sizeof(uint64_t)));
	CudaSafeCall(hipHostAlloc(&_2GnxPinned, 4 * sizeof(uint64_t), hipHostMallocWriteCombined | hipHostMallocMapped));

	CudaSafeCall(hipMalloc((void**)&__2Gny, 4 * sizeof(uint64_t)));
	CudaSafeCall(hipHostAlloc(&_2GnyPinned, 4 * sizeof(uint64_t), hipHostMallocWriteCombined | hipHostMallocMapped));

	size_t TSIZE = (size / 2) * 4 * sizeof(uint64_t);
	CudaSafeCall(hipMalloc((void**)&_Gx, TSIZE));
	CudaSafeCall(hipHostAlloc(&GxPinned, TSIZE, hipHostMallocWriteCombined | hipHostMallocMapped));

	CudaSafeCall(hipMalloc((void**)&_Gy, TSIZE));
	CudaSafeCall(hipHostAlloc(&GyPinned, TSIZE, hipHostMallocWriteCombined | hipHostMallocMapped));


	Point* Gn = new Point[size];
	Point g = secp->G;
	Gn[0] = g;
	g = secp->DoubleDirect(g);
	Gn[1] = g;
	for (int i = 2; i < size; i++) {
		g = secp->AddDirect(g, secp->G);
		Gn[i] = g;
	}
	// _2Gn = CPU_GRP_SIZE*G
	Point _2Gn = secp->DoubleDirect(Gn[size / 2 - 1]);

	int nbDigit = 4;
	for (int i = 0; i < nbDigit; i++) {
		_2GnxPinned[i] = _2Gn.x.bits64[i];
		_2GnyPinned[i] = _2Gn.y.bits64[i];
	}
	for (int i = 0; i < size / 2; i++) {
		for (int j = 0; j < nbDigit; j++) {
			GxPinned[i * nbDigit + j] = Gn[i].x.bits64[j];
			GyPinned[i * nbDigit + j] = Gn[i].y.bits64[j];
		}
	}

	delete[] Gn;

	CudaSafeCall(hipMemcpy(__2Gnx, _2GnxPinned, 4 * sizeof(uint64_t), hipMemcpyHostToDevice));
	CudaSafeCall(hipHostFree(_2GnxPinned));
	_2GnxPinned = NULL;

	CudaSafeCall(hipMemcpy(__2Gny, _2GnyPinned, 4 * sizeof(uint64_t), hipMemcpyHostToDevice));
	CudaSafeCall(hipHostFree(_2GnyPinned));
	_2GnyPinned = NULL;

	CudaSafeCall(hipMemcpy(_Gx, GxPinned, TSIZE, hipMemcpyHostToDevice));
	CudaSafeCall(hipHostFree(GxPinned));
	GxPinned = NULL;

	CudaSafeCall(hipMemcpy(_Gy, GyPinned, TSIZE, hipMemcpyHostToDevice));
	CudaSafeCall(hipHostFree(GyPinned));
	GyPinned = NULL;

	CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(_2Gnx), &__2Gnx, sizeof(uint64_t*)));
	CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(_2Gny), &__2Gny, sizeof(uint64_t*)));
	CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(Gx), &_Gx, sizeof(uint64_t*)));
	CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(Gy), &_Gy, sizeof(uint64_t*)));

}

// ----------------------------------------------------------------------------

int GPUEngine::GetGroupSize()
{
	return GRP_SIZE;
}

// ----------------------------------------------------------------------------

void GPUEngine::PrintCudaInfo()
{
	const char* sComputeMode[] = {
		"Multiple host threads",
		"Only one host thread",
		"No host thread",
		"Multiple process threads",
		"Unknown",
		NULL
	};

	int deviceCount = 0;
	CudaSafeCall(hipGetDeviceCount(&deviceCount));

	// This function call returns 0 if there are no CUDA capable devices.
	if (deviceCount == 0) {
		printf("GPUEngine: There are no available device(s) that support CUDA\n");
		return;
	}

	for (int i = 0; i < deviceCount; i++) {
		CudaSafeCall(hipSetDevice(i));
		hipDeviceProp_t deviceProp;
		CudaSafeCall(hipGetDeviceProperties(&deviceProp, i));
		printf("GPU #%d %s (%dx%d cores) (Cap %d.%d) (%.1f MB) (%s)\n",
			i, deviceProp.name, deviceProp.multiProcessorCount,
			_ConvertSMVer2Cores(deviceProp.major, deviceProp.minor),
			deviceProp.major, deviceProp.minor, (double)deviceProp.totalGlobalMem / 1048576.0,
			sComputeMode[deviceProp.computeMode]);
	}
}

// ----------------------------------------------------------------------------

GPUEngine::~GPUEngine()
{
	CudaSafeCall(hipFree(inputKey));
	if (searchMode == (int)SEARCH_MODE_MA || searchMode == (int)SEARCH_MODE_MX)
		CudaSafeCall(hipFree(inputBloomLookUp));
	else
		CudaSafeCall(hipFree(inputHashORxpoint));

	CudaSafeCall(hipHostFree(outputBufferPinned));
	CudaSafeCall(hipFree(outputBuffer));

	CudaSafeCall(hipFree(__2Gnx));
	CudaSafeCall(hipFree(__2Gny));
	CudaSafeCall(hipFree(_Gx));
	CudaSafeCall(hipFree(_Gy));

	if (rKey)
		CudaSafeCall(hipHostFree(inputKeyPinned));
}

// ----------------------------------------------------------------------------

int GPUEngine::GetNbThread()
{
	return nbThread;
}

// ----------------------------------------------------------------------------

bool GPUEngine::callKernelSEARCH_MODE_MA()
{

	// Reset nbFound
	CudaSafeCall(hipMemset(outputBuffer, 0, 4));

	// Call the kernel (Perform STEP_SIZE keys per thread)
	if (coinType == COIN_BTC) {
		if (compMode == SEARCH_COMPRESSED) {
			compute_keys_comp_mode_ma << < nbThread / nbThreadPerGroup, nbThreadPerGroup >> >
				(compMode, inputBloomLookUp, BLOOM_BITS, BLOOM_HASHES, inputKey, maxFound, outputBuffer);
		}
		else {
			compute_keys_mode_ma << < nbThread / nbThreadPerGroup, nbThreadPerGroup >> >
				(compMode, inputBloomLookUp, BLOOM_BITS, BLOOM_HASHES, inputKey, maxFound, outputBuffer);
		}
	}
	else {
		compute_keys_mode_eth_ma << < nbThread / nbThreadPerGroup, nbThreadPerGroup >> >
			(inputBloomLookUp, BLOOM_BITS, BLOOM_HASHES, inputKey, maxFound, outputBuffer);
	}

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		printf("GPUEngine: Kernel: %s\n", hipGetErrorString(err));
		return false;
	}
	return true;

}

// ----------------------------------------------------------------------------

bool GPUEngine::callKernelSEARCH_MODE_MX()
{

	// Reset nbFound
	CudaSafeCall(hipMemset(outputBuffer, 0, 4));

	// Call the kernel (Perform STEP_SIZE keys per thread)
	if (compMode == SEARCH_COMPRESSED) {
		compute_keys_comp_mode_mx << < nbThread / nbThreadPerGroup, nbThreadPerGroup >> >
			(compMode, inputBloomLookUp, BLOOM_BITS, BLOOM_HASHES, inputKey, maxFound, outputBuffer);
	}
	else {
		printf("GPUEngine: PubKeys search doesn't support uncompressed\n");
		return false;
	}

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		printf("GPUEngine: Kernel: %s\n", hipGetErrorString(err));
		return false;
	}
	return true;
}

// ----------------------------------------------------------------------------

bool GPUEngine::callKernelSEARCH_MODE_SA()
{

	// Reset nbFound
	CudaSafeCall(hipMemset(outputBuffer, 0, 4));

	// Call the kernel (Perform STEP_SIZE keys per thread)
	if (coinType == COIN_BTC) {
		if (compMode == SEARCH_COMPRESSED) {
			compute_keys_comp_mode_sa << < nbThread / nbThreadPerGroup, nbThreadPerGroup >> >
				(compMode, inputHashORxpoint, inputKey, maxFound, outputBuffer);
		}
		else {
			compute_keys_mode_sa << < nbThread / nbThreadPerGroup, nbThreadPerGroup >> >
				(compMode, inputHashORxpoint, inputKey, maxFound, outputBuffer);
		}
	}
	else {
		compute_keys_mode_eth_sa << < nbThread / nbThreadPerGroup, nbThreadPerGroup >> >
			(inputHashORxpoint, inputKey, maxFound, outputBuffer);
	}

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		printf("GPUEngine: Kernel: %s\n", hipGetErrorString(err));
		return false;
	}
	return true;

}

// ----------------------------------------------------------------------------

bool GPUEngine::callKernelSEARCH_MODE_SX()
{

	// Reset nbFound
	CudaSafeCall(hipMemset(outputBuffer, 0, 4));

	// Call the kernel (Perform STEP_SIZE keys per thread)
	if (compMode == SEARCH_COMPRESSED) {
		compute_keys_comp_mode_sx << < nbThread / nbThreadPerGroup, nbThreadPerGroup >> >
			(compMode, inputHashORxpoint, inputKey, maxFound, outputBuffer);
	}
	else {
		printf("GPUEngine: PubKeys search doesn't support uncompressed\n");
		return false;
	}

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		printf("GPUEngine: Kernel: %s\n", hipGetErrorString(err));
		return false;
	}
	return true;
}

// ----------------------------------------------------------------------------

bool GPUEngine::SetKeys(Point* p)
{
	// Sets the starting keys for each thread
	// p must contains nbThread public keys
	for (int i = 0; i < nbThread; i += nbThreadPerGroup) {
		for (int j = 0; j < nbThreadPerGroup; j++) {

			inputKeyPinned[8 * i + j + 0 * nbThreadPerGroup] = p[i + j].x.bits64[0];
			inputKeyPinned[8 * i + j + 1 * nbThreadPerGroup] = p[i + j].x.bits64[1];
			inputKeyPinned[8 * i + j + 2 * nbThreadPerGroup] = p[i + j].x.bits64[2];
			inputKeyPinned[8 * i + j + 3 * nbThreadPerGroup] = p[i + j].x.bits64[3];

			inputKeyPinned[8 * i + j + 4 * nbThreadPerGroup] = p[i + j].y.bits64[0];
			inputKeyPinned[8 * i + j + 5 * nbThreadPerGroup] = p[i + j].y.bits64[1];
			inputKeyPinned[8 * i + j + 6 * nbThreadPerGroup] = p[i + j].y.bits64[2];
			inputKeyPinned[8 * i + j + 7 * nbThreadPerGroup] = p[i + j].y.bits64[3];

		}
	}

	// Fill device memory
	CudaSafeCall(hipMemcpy(inputKey, inputKeyPinned, nbThread * 32 * 2, hipMemcpyHostToDevice));

	if (!rKey) {
		// We do not need the input pinned memory anymore
		CudaSafeCall(hipHostFree(inputKeyPinned));
		inputKeyPinned = NULL;
	}

	switch (searchMode) {
	case (int)SEARCH_MODE_MA:
		return callKernelSEARCH_MODE_MA();
		break;
	case (int)SEARCH_MODE_SA:
		return callKernelSEARCH_MODE_SA();
		break;
	case (int)SEARCH_MODE_MX:
		return callKernelSEARCH_MODE_MX();
		break;
	case (int)SEARCH_MODE_SX:
		return callKernelSEARCH_MODE_SX();
		break;
	default:
		return false;
		break;
	}
}

// ----------------------------------------------------------------------------

bool GPUEngine::LaunchSEARCH_MODE_MA(std::vector<ITEM>& dataFound, bool spinWait)
{

	dataFound.clear();

	// Get the result
	if (spinWait) {
		CudaSafeCall(hipMemcpy(outputBufferPinned, outputBuffer, outputSize, hipMemcpyDeviceToHost));
	}
	else {
		// Use hipMemcpyAsync to avoid default spin wait of hipMemcpy wich takes 100% CPU
		hipEvent_t evt;
		CudaSafeCall(hipEventCreate(&evt));
		CudaSafeCall(hipMemcpyAsync(outputBufferPinned, outputBuffer, 4, hipMemcpyDeviceToHost, 0));
		CudaSafeCall(hipEventRecord(evt, 0));
		while (hipEventQuery(evt) == hipErrorNotReady) {
			// Sleep 1 ms to free the CPU
			Timer::SleepMillis(1);
		}
		CudaSafeCall(hipEventDestroy(evt));
	}

	// Look for data found
	uint32_t nbFound = outputBufferPinned[0];
	if (nbFound > maxFound) {
		nbFound = maxFound;
	}

	// When can perform a standard copy, the kernel is eneded
	CudaSafeCall(hipMemcpy(outputBufferPinned, outputBuffer, nbFound * ITEM_SIZE_A + 4, hipMemcpyDeviceToHost));

	for (uint32_t i = 0; i < nbFound; i++) {

		uint32_t* itemPtr = outputBufferPinned + (i * ITEM_SIZE_A32 + 1);
		uint8_t* hash = (uint8_t*)(itemPtr + 2);
		if (CheckBinary(hash, 20) > 0) {

			ITEM it;
			it.thId = itemPtr[0];
			int16_t* ptr = (int16_t*)&(itemPtr[1]);
			//it.endo = ptr[0] & 0x7FFF;
			it.mode = (ptr[0] & 0x8000) != 0;
			it.incr = ptr[1];
			it.hash = (uint8_t*)(itemPtr + 2);
			dataFound.push_back(it);
		}
	}
	return callKernelSEARCH_MODE_MA();
}

// ----------------------------------------------------------------------------

bool GPUEngine::LaunchSEARCH_MODE_SA(std::vector<ITEM>& dataFound, bool spinWait)
{

	dataFound.clear();

	// Get the result
	if (spinWait) {
		CudaSafeCall(hipMemcpy(outputBufferPinned, outputBuffer, outputSize, hipMemcpyDeviceToHost));
	}
	else {
		// Use hipMemcpyAsync to avoid default spin wait of hipMemcpy wich takes 100% CPU
		hipEvent_t evt;
		CudaSafeCall(hipEventCreate(&evt));
		CudaSafeCall(hipMemcpyAsync(outputBufferPinned, outputBuffer, 4, hipMemcpyDeviceToHost, 0));
		CudaSafeCall(hipEventRecord(evt, 0));
		while (hipEventQuery(evt) == hipErrorNotReady) {
			// Sleep 1 ms to free the CPU
			Timer::SleepMillis(1);
		}
		CudaSafeCall(hipEventDestroy(evt));
	}

	// Look for data found
	uint32_t nbFound = outputBufferPinned[0];
	if (nbFound > maxFound) {
		nbFound = maxFound;
	}

	// When can perform a standard copy, the kernel is eneded
	CudaSafeCall(hipMemcpy(outputBufferPinned, outputBuffer, nbFound * ITEM_SIZE_A + 4, hipMemcpyDeviceToHost));

	for (uint32_t i = 0; i < nbFound; i++) {
		uint32_t* itemPtr = outputBufferPinned + (i * ITEM_SIZE_A32 + 1);
		ITEM it;
		it.thId = itemPtr[0];
		int16_t* ptr = (int16_t*)&(itemPtr[1]);
		//it.endo = ptr[0] & 0x7FFF;
		it.mode = (ptr[0] & 0x8000) != 0;
		it.incr = ptr[1];
		it.hash = (uint8_t*)(itemPtr + 2);
		dataFound.push_back(it);
	}
	return callKernelSEARCH_MODE_SA();
}

// ----------------------------------------------------------------------------

bool GPUEngine::LaunchSEARCH_MODE_MX(std::vector<ITEM>& dataFound, bool spinWait)
{

	dataFound.clear();

	// Get the result
	if (spinWait) {
		CudaSafeCall(hipMemcpy(outputBufferPinned, outputBuffer, outputSize, hipMemcpyDeviceToHost));
	}
	else {
		// Use hipMemcpyAsync to avoid default spin wait of hipMemcpy wich takes 100% CPU
		hipEvent_t evt;
		CudaSafeCall(hipEventCreate(&evt));
		CudaSafeCall(hipMemcpyAsync(outputBufferPinned, outputBuffer, 4, hipMemcpyDeviceToHost, 0));
		CudaSafeCall(hipEventRecord(evt, 0));
		while (hipEventQuery(evt) == hipErrorNotReady) {
			// Sleep 1 ms to free the CPU
			Timer::SleepMillis(1);
		}
		CudaSafeCall(hipEventDestroy(evt));
	}

	// Look for data found
	uint32_t nbFound = outputBufferPinned[0];
	if (nbFound > maxFound) {
		nbFound = maxFound;
	}

	// When can perform a standard copy, the kernel is eneded
	CudaSafeCall(hipMemcpy(outputBufferPinned, outputBuffer, nbFound * ITEM_SIZE_X + 4, hipMemcpyDeviceToHost));

	for (uint32_t i = 0; i < nbFound; i++) {

		uint32_t* itemPtr = outputBufferPinned + (i * ITEM_SIZE_X32 + 1);
		uint8_t* pubkey = (uint8_t*)(itemPtr + 2);

		if (CheckBinary(pubkey, 32) > 0) {

			ITEM it;
			it.thId = itemPtr[0];
			int16_t* ptr = (int16_t*)&(itemPtr[1]);
			//it.endo = ptr[0] & 0x7FFF;
			it.mode = (ptr[0] & 0x8000) != 0;
			it.incr = ptr[1];
			it.hash = (uint8_t*)(itemPtr + 2);
			dataFound.push_back(it);
		}
	}
	return callKernelSEARCH_MODE_MX();
}

// ----------------------------------------------------------------------------

bool GPUEngine::LaunchSEARCH_MODE_SX(std::vector<ITEM>& dataFound, bool spinWait)
{

	dataFound.clear();

	// Get the result
	if (spinWait) {
		CudaSafeCall(hipMemcpy(outputBufferPinned, outputBuffer, outputSize, hipMemcpyDeviceToHost));
	}
	else {
		// Use hipMemcpyAsync to avoid default spin wait of hipMemcpy wich takes 100% CPU
		hipEvent_t evt;
		CudaSafeCall(hipEventCreate(&evt));
		CudaSafeCall(hipMemcpyAsync(outputBufferPinned, outputBuffer, 4, hipMemcpyDeviceToHost, 0));
		CudaSafeCall(hipEventRecord(evt, 0));
		while (hipEventQuery(evt) == hipErrorNotReady) {
			// Sleep 1 ms to free the CPU
			Timer::SleepMillis(1);
		}
		CudaSafeCall(hipEventDestroy(evt));
	}

	// Look for data found
	uint32_t nbFound = outputBufferPinned[0];
	if (nbFound > maxFound) {
		nbFound = maxFound;
	}

	// When can perform a standard copy, the kernel is eneded
	CudaSafeCall(hipMemcpy(outputBufferPinned, outputBuffer, nbFound * ITEM_SIZE_X + 4, hipMemcpyDeviceToHost));

	for (uint32_t i = 0; i < nbFound; i++) {

		uint32_t* itemPtr = outputBufferPinned + (i * ITEM_SIZE_X32 + 1);
		uint8_t* pubkey = (uint8_t*)(itemPtr + 2);

		ITEM it;
		it.thId = itemPtr[0];
		int16_t* ptr = (int16_t*)&(itemPtr[1]);
		//it.endo = ptr[0] & 0x7FFF;
		it.mode = (ptr[0] & 0x8000) != 0;
		it.incr = ptr[1];
		it.hash = (uint8_t*)(itemPtr + 2);
		dataFound.push_back(it);
	}
	return callKernelSEARCH_MODE_SX();
}

// ----------------------------------------------------------------------------

int GPUEngine::CheckBinary(const uint8_t* _x, int K_LENGTH)
{
	uint8_t* temp_read;
	uint64_t half, min, max, current; //, current_offset
	int64_t rcmp;
	int32_t r = 0;
	min = 0;
	current = 0;
	max = TOTAL_COUNT;
	half = TOTAL_COUNT;
	while (!r && half >= 1) {
		half = (max - min) / 2;
		temp_read = DATA + ((current + half) * K_LENGTH);
		rcmp = memcmp(_x, temp_read, K_LENGTH);
		if (rcmp == 0) {
			r = 1;  //Found!!
		}
		else {
			if (rcmp < 0) { //data < temp_read
				max = (max - half);
			}
			else { // data > temp_read
				min = (min + half);
			}
			current = min;
		}
	}
	return r;
}




